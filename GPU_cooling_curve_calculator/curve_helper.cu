#include "hip/hip_runtime.h"
#include ""
#include "gpu.h"

#define LinearInterpolateCalc(x0, x1, y0, y1, x) (y0 + ((y1 - y0) * ((x - x0) / PREC(x1 - x0))))

__device__ PREC
LinearInterpolationWithFixDx
                    (
                      const PREC* _x_values,
                      const PREC* _y_values,
                      const int _N,
                      const PREC _x
                    )
{
  assume( ( _x >= _x_values[ 0 ] ) && ( _x <= _x_values[ _N - 1 ] ), "_x:%lf is out ouf bounds", _x );

  const PREC c1pdx = 1 / (_x_values[1] - _x_values[0]);
  const PREC x_parcel = c1pdx * (_x - _x_values[0]);

  const int index = __float2int_rz(x_parcel);

  const PREC yp = _y_values[ index ];
  const PREC yp1 = _y_values[ index + 1];

  const PREC x_weight = x_parcel - truncf(x_parcel);
  const PREC y = (1.f - x_weight) * yp + x_weight * yp1;

  assume( ((  y == yp ) && (  y == yp1 ))|| ((  y >= yp ) && (  y <= yp1 )) || ((  y >= yp1 ) && (  y <= yp )), "y:%lf is out ouf bounds; yp:%lf; yp1:%lf", y, yp, yp1 );
  return ((yp == yp1) ? yp : y);
}


__device__ PREC
CalculateK(PREC T)
{
  return PRECL(13.7129) + PRECL(0.017) * T;
}

__device__ PREC
CalculateCp(PREC T)
{
  return PRECL(333.73) + PRECL(0.2762) * T;
}

__device__ PREC
CalculateRho(PREC T)
{
  return PRECL(7925.4) - PRECL(0.4434) * T;
}

__device__ PREC
CalculateAlpha(PREC _temperature)
{
  PREC k = CalculateK(_temperature);
  PREC cp = CalculateCp(_temperature);
  PREC rho = CalculateRho(_temperature);

  return k  / (cp * rho);
}

PREC
H_CalculateK(PREC T)
{
  return PRECL(13.7129) + PRECL(0.017) * T;
}

PREC
H_CalculateCp(PREC T)
{
  return PRECL(333.73) + PRECL(0.2762) * T;
}

PREC
H_CalculateRho(PREC T)
{
  return PRECL(7925.4) - PRECL(0.4434) * T;
}

PREC
H_CalculateAlpha(PREC _temperature)
{
  PREC k = H_CalculateK(_temperature);
  PREC cp = H_CalculateCp(_temperature);
  PREC rho = H_CalculateRho(_temperature);

  return k / (cp * rho);
}