#include "hip/hip_runtime.h"
#include <sys/time.h>

#include ""
#include "gpu.h"
#include "curve_helper.cu"
#include "io_helper.c"

#define WARP_SIZE 32
#define SHM_MAX_MAX (48 * 1024)
#define SHM_MAX_MAX_FOR_ALLOC (12 * 1024)



__constant__ GPUConfig d_config;

__shared__ int TCIndex;
__shared__ int lastTemperatureIndex;
__shared__ PREC dtpdx2;
__shared__ PREC dtdx2m2;
__shared__ int htc_len;
__shared__ int cooling_curve_len;
__shared__ PREC temperature_final;
__shared__ int temperatures_len;
__shared__ PREC temperature_init;
__shared__ PREC end_time;
__shared__ PREC delta_t;
__shared__ int interpolated_htc_by_dt_len;
__shared__ int calculation_count;



inline int
warponizing(int _num)
{
  return (int)ceil( _num / (PREC)WARP_SIZE ) * WARP_SIZE;
}

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, char const* const func, char const* const file,
           int const line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(char const* const file, int const line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

__device__ void
FillArrayAll(PREC* const _array, const PREC _init_value, const int _len)
{
  const int cicle = _len / blockDim.x;
  const int filled_items = cicle * blockDim.x;

  for ( int i = 0; i < cicle; i++ )
  {
    const int array_index = threadIdx.x + i * blockDim.x;
    _array[ array_index ] = _init_value;
  }

  if ( threadIdx.x < _len - filled_items )
  {
    const int array_index = filled_items + threadIdx.x;
    _array[ array_index ] = _init_value;
  }

  __syncthreads();
}

__device__ void
CopyArrayAll(PREC* const _dst, const PREC* const _src, const int _len)
{
  const int cicle = _len / blockDim.x;
  const int copied_items = cicle * blockDim.x;

  for ( int i = 0; i < cicle; i++ )
  {
    const int array_index = threadIdx.x + i * blockDim.x;
    _dst[ array_index ] = _src[ array_index ];
  }

  if ( threadIdx.x < _len - copied_items )
  {
    const int array_index = copied_items + threadIdx.x;
    _dst[ array_index ] = _src[ array_index ];
  }

  __syncthreads();
}

__device__ void
CalculateCooling_curve(
                       PREC* _temperatures,
                       PREC* _nextTemperatures,
                       PREC _decTmodmHTC,
                       PREC* _c1pc1p2i,
                       PREC* _c1mc1p2i
                      )
{
  if ( threadIdx.x > WARP_SIZE && threadIdx.x < lastTemperatureIndex + WARP_SIZE )
  {
    const int idx = threadIdx.x - WARP_SIZE;
    const PREC temp_i = _temperatures[ idx ];
    const PREC temp_im1 = _temperatures[ idx - 1 ];
    const PREC temp_ip1 = _temperatures[ idx + 1 ];
    const PREC alpha = CalculateAlpha( temp_i );
    const PREC c1pc1p2i = _c1pc1p2i[ idx ];
    const PREC c1mc1p2i = _c1mc1p2i[ idx ];

    _nextTemperatures[ idx ] = temp_i + (alpha * dtpdx2) * ( temp_ip1 * c1pc1p2i + temp_im1 * c1mc1p2i - 2 * temp_i );
  }

  if ( threadIdx.x == 0 )
  {
    const PREC temp_0 = _temperatures[ 0 ];
    const PREC temp_1 = _temperatures[ 1 ];
    const PREC alpha = CalculateAlpha( temp_0 );

    _nextTemperatures[ 0 ] = temp_0 + (alpha * dtdx2m2) * ( temp_1 - temp_0 );
  }

  if ( threadIdx.x == lastTemperatureIndex + WARP_SIZE )
  {
    const PREC temp_last = _temperatures[ lastTemperatureIndex ];
    const PREC temp_lastm1 = _temperatures[ lastTemperatureIndex - 1 ];
    const PREC alpha = CalculateAlpha( temp_last );
    const PREC c1pK_last_m2 = 1 / (2 * CalculateK( temp_last ));
    const PREC decrement_T  = ( temp_last - temperature_final ) * _decTmodmHTC * c1pK_last_m2;

    _nextTemperatures[ lastTemperatureIndex ] = temp_last + (alpha * dtdx2m2) * ( temp_lastm1 - temp_last - decrement_T );
  }
}

__device__ void
StoreData(
                PREC*  const _cooling_curve_values,
                const PREC _value,
                int* const _store_index
               )
{
  _cooling_curve_values[ *_store_index ] = _value;
  (*_store_index)++;
}

__device__ void
interpolated_htc_by_dt(
             PREC* const _interpolated_dt_htc_value,
       const PREC* const _htc_times,
       const PREC* const _htc_values,
             int* const _start
       )
{
  const int len = min( interpolated_htc_by_dt_len, (calculation_count - *_start));
  const int cicle = len / blockDim.x;
  const int calculated_items = cicle * blockDim.x;

  for ( int i = 0; i < cicle; i++ )
  {
    const int index = threadIdx.x + i * blockDim.x;
    const PREC time = delta_t * (index + *_start);
    _interpolated_dt_htc_value[ index ] = LinearInterpolationWithFixDx( _htc_times, _htc_values, htc_len, time );
  }


  if ( threadIdx.x < len - calculated_items )
  {
    const int index = calculated_items + threadIdx.x;
    const PREC time = delta_t * (index + *_start);
    _interpolated_dt_htc_value[ index ] = LinearInterpolationWithFixDx( _htc_times, _htc_values, htc_len, time );
  }

  *_start += len;
  __syncthreads();
}

__global__ void
CalculateCooling_curveWorker(
                            const int _number_of_curve,
                            const int _interpolated_htc_by_dt_len,
                            const int _calculation_count,
                            const PREC  _decTmod,
                            const PREC  _dtpdx2,
                            const PREC  _dtdx2m2,
                            const PREC  _delta_t,
                                  PREC* const _g_htc_values,
                            const PREC* const _g_htc_times,
                                  PREC* const _g_cooling_curve_values,
                            const PREC* const _g_cooling_curve_times,
                            const PREC* const _g_c1pc1p2i,
                            const PREC* const _g_c1mc1p2i
                           )
{
  if ( blockIdx.x < _number_of_curve )
  {
    extern __shared__ __volatile__ PREC dyn_share_memory[];

    if (threadIdx.x == 0)
    {
      end_time = d_config.end_time;
      htc_len = d_config.htc_N;
      cooling_curve_len = d_config.cooling_curve_N;
      temperatures_len = d_config.N;
      temperature_init = d_config.initial_temperature;
      temperature_final = d_config.final_temperature;
      lastTemperatureIndex = d_config.lastTemperatureIndex;
      TCIndex = d_config.TCIndex;
      delta_t = _delta_t;
      interpolated_htc_by_dt_len = _interpolated_htc_by_dt_len;
      calculation_count = _calculation_count;
      dtpdx2 = _dtpdx2;
      dtdx2m2 = _dtdx2m2;
    }
    __syncthreads();

    PREC* htc_values = (PREC*)&dyn_share_memory;
    PREC* htc_times = htc_values + htc_len;
    PREC* cooling_curve_values = htc_times + htc_len;
    PREC* cooling_curve_times = cooling_curve_values + cooling_curve_len;
    PREC* temperatures = cooling_curve_times + cooling_curve_len;
    PREC* nextTemperatures = temperatures + temperatures_len;
    PREC* c1pc1p2i = nextTemperatures + temperatures_len;
    PREC* c1mc1p2i = c1pc1p2i + temperatures_len;
    PREC* interpolated_htc_by_dt_value = c1mc1p2i + temperatures_len;

    PREC time = 0;
    PREC reference_time = 0;
    int ref_time_index = 0;
    int store_index = 0;
    int time_index = 0;
    int interpolated_htc_by_dt_next_start = 0;
    int interpolated_htc_by_dt_index = 0;

    PREC* htc_values_source = _g_htc_values + blockIdx.x * htc_len;
    PREC* cooling_curve_values_source = _g_cooling_curve_values + blockIdx.x * cooling_curve_len;

    CopyArrayAll(htc_values, htc_values_source, htc_len);
    CopyArrayAll(htc_times, _g_htc_times, htc_len);
    CopyArrayAll(cooling_curve_times, _g_cooling_curve_times, cooling_curve_len);
    CopyArrayAll(c1pc1p2i, _g_c1pc1p2i, temperatures_len);
    CopyArrayAll(c1mc1p2i, _g_c1mc1p2i, temperatures_len);

    FillArrayAll(temperatures, temperature_init, temperatures_len);

    interpolated_htc_by_dt(interpolated_htc_by_dt_value, htc_times, htc_values, &interpolated_htc_by_dt_next_start);

    StoreData(
               cooling_curve_values,
               temperatures[ TCIndex ],
               &store_index
              );

    ++time_index;
    ++ref_time_index;
    ++interpolated_htc_by_dt_index;
    time = delta_t;
    reference_time = cooling_curve_times[ ref_time_index ];

    while ( time < end_time )
    {
      const PREC htc = interpolated_htc_by_dt_value[ interpolated_htc_by_dt_index ];

      CalculateCooling_curve(
                            temperatures,
                            nextTemperatures,
                            _decTmod * htc,
                            c1pc1p2i, c1mc1p2i
                           );

      if ( time > reference_time )
      {
        const PREC prev_time = time - delta_t;
        const PREC prev_temperature = temperatures[ TCIndex ];
        const PREC temperature = nextTemperatures[ TCIndex ];
        const PREC reftime_temperature =
                   LinearInterpolateCalc(prev_time, time, prev_temperature, temperature, reference_time);
        StoreData(cooling_curve_values, reftime_temperature, &store_index);

        ++ref_time_index;
        reference_time = ref_time_index < cooling_curve_len ? cooling_curve_times[ ref_time_index ]: end_time;
      }
      SWAP( temperatures, nextTemperatures );

      ++interpolated_htc_by_dt_index;
      if ( interpolated_htc_by_dt_index == interpolated_htc_by_dt_len )
      {
        __syncthreads();
        interpolated_htc_by_dt_index = 0;
        interpolated_htc_by_dt(interpolated_htc_by_dt_value, htc_times, htc_values, &interpolated_htc_by_dt_next_start);
      }

      ++time_index;
      time = delta_t * time_index;
    }

    const PREC htc = interpolated_htc_by_dt_value[ interpolated_htc_by_dt_index ];

    CalculateCooling_curve(
                        temperatures,
                        nextTemperatures,
                        _decTmod * htc,
                        c1pc1p2i, c1mc1p2i
                       );

    StoreData(
               cooling_curve_values,
               nextTemperatures[ TCIndex ],
               &store_index
              );

    __syncthreads();
    CopyArrayAll(cooling_curve_values_source, cooling_curve_values, cooling_curve_len);
  }
}

static void
CalculateWorker(const int _number_of_curve, Config* config)
{
  struct timeval tval_before, tval_after;
  gettimeofday(&tval_before, NULL);

  PREC* d_htc_values;
  PREC* d_htc_times;
  PREC* d_cooling_curve_values;
  PREC* d_cooling_curve_times;

  PREC* d_c1pc1p2i;
  PREC* d_c1mc1p2i;

  GPUConfig gpuconfig;

  const int lastTemperatureIndex = config->N - 1;

  gpuconfig.htc_N = config->htc_N;
  gpuconfig.end_time = config->end_time;
  gpuconfig.initial_temperature = config->initial_temperature;
  gpuconfig.final_temperature = config->final_temperature;
  gpuconfig.R = config->R;
  gpuconfig.N = config->N;
  gpuconfig.TC = config->TC;
  gpuconfig.cooling_curve_N = config->cooling_curve_N;
  gpuconfig.lastTemperatureIndex = lastTemperatureIndex;
  gpuconfig.TCIndex = config->TC - 1;

  const PREC dx = config->R / (PREC)config->N;
  const PREC dx2 = dx * dx;
  const PREC c1pdx2 = 1.0f / dx2;
  const PREC c2mdx2 = 2.0f * dx2;
  const PREC dx4 = dx2 * dx2;
  const PREC delta_t = dx2 / 2.0f / H_CalculateAlpha(config->initial_temperature); //stability_dt
  const PREC decTmod = ( 2.0f + 1.0f / ( (PREC)lastTemperatureIndex * dx4 ) ) * dx;
  const PREC dtpdx2 = delta_t * c1pdx2;
  const PREC dtdx2m2 = delta_t * c2mdx2;
  const PREC calculation_count = (int)floor( config->end_time / delta_t + 1);

  config->c1pc1p2i[ 0 ] = 1;
  config->c1mc1p2i[ 0 ] = 1;
  for (int i = 1; i < config->N; ++i) // divide by zero if i=0
  {
    const PREC c1p2i = 1.0f / ( (PREC)(i << 1) );
    config->c1pc1p2i[ i ] = 1 + c1p2i;
    config->c1mc1p2i[ i ] = 1 - c1p2i;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  CHECK_CUDA_ERROR(hipMemcpyToSymbol((const void*)&d_config, &gpuconfig, sizeof(GPUConfig)));

  CHECK_CUDA_ERROR(hipMalloc((void**)&d_cooling_curve_times, config->cooling_curve_N * sizeof(PREC)));
  CHECK_CUDA_ERROR(hipMemcpy((void*)d_cooling_curve_times, (const void *)config->cooling_curve_times, config->cooling_curve_N * sizeof(PREC), hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(hipMalloc((void**)&d_htc_values, _number_of_curve * config->htc_N * sizeof(PREC)));
  CHECK_CUDA_ERROR(hipMemcpy((void*)d_htc_values, (const void *)config->htc_values, _number_of_curve * config->htc_N * sizeof(PREC), hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(hipMalloc((void**)&d_htc_times, config->htc_N * sizeof(PREC)));
  CHECK_CUDA_ERROR(hipMemcpy((void*)d_htc_times, (const void *)config->htc_times, config->htc_N * sizeof(PREC), hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(hipMalloc((void**)&d_c1pc1p2i, config->N * sizeof(PREC)));
  CHECK_CUDA_ERROR(hipMemcpy((void*)d_c1pc1p2i, (const void *)config->c1pc1p2i, config->N * sizeof(PREC), hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(hipMalloc((void**)&d_c1mc1p2i, config->N * sizeof(PREC)));
  CHECK_CUDA_ERROR(hipMemcpy((void*)d_c1mc1p2i, (const void *)config->c1mc1p2i, config->N * sizeof(PREC), hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(hipMalloc((void**)&d_cooling_curve_values, _number_of_curve * config->cooling_curve_N * sizeof(PREC)));

  dim3 threadNum = dim3( warponizing( config->N + 64), 1 );                        //grid_width
  dim3 blockNum =  dim3( _number_of_curve, 1);                                     //number of curve

  const int shared_mem_fixed_data_count =
                       (
                        config->htc_N // htc_values
                        +
                        config->htc_N // htc_times
                        +
                        config->cooling_curve_N //cooling_curve_values
                        +
                        config->cooling_curve_N //cooling_curve_times
                        +
                        config->N // temperatures
                        +
                        config->N // next_temperatures
                        +
                        config->N // c1pc1p2i
                        +
                        config->N // c1mc1p2i
                        // +
                        // config->cooling_curve_N // interpolated_htc_by_reftime
                       );

  const int interpolated_htc_by_dt_len = SHM_MAX_MAX_FOR_ALLOC / sizeof(PREC) - shared_mem_fixed_data_count;
  assume( interpolated_htc_by_dt_len > 0, "interpolated_htc_by_dt_len:%d is not pozitive", interpolated_htc_by_dt_len);

  int shared_mem_len = SHM_MAX_MAX_FOR_ALLOC;

  hipEventRecord(start);

  CalculateCooling_curveWorker <<< blockNum, threadNum, shared_mem_len >>> (
                                                                            _number_of_curve,
                                                                            interpolated_htc_by_dt_len,
                                                                            calculation_count,
                                                                            decTmod,
                                                                            dtpdx2,
                                                                            dtdx2m2,
                                                                            delta_t,
                                                                            d_htc_values,
                                                                            d_htc_times,
                                                                            d_cooling_curve_values,
                                                                            d_cooling_curve_times,
                                                                            d_c1pc1p2i,
                                                                            d_c1mc1p2i
                                                                           );

  CHECK_LAST_CUDA_ERROR();

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  CHECK_CUDA_ERROR(hipDeviceSynchronize());

  CHECK_CUDA_ERROR(hipMemcpy((void*)config->cooling_curve_values, (const void *)d_cooling_curve_values, _number_of_curve * config->cooling_curve_N * sizeof(PREC), hipMemcpyDeviceToHost));


  CHECK_CUDA_ERROR(hipFree(d_cooling_curve_values));
  CHECK_CUDA_ERROR(hipFree(d_cooling_curve_times));
  CHECK_CUDA_ERROR(hipFree(d_htc_values));
  CHECK_CUDA_ERROR(hipFree(d_htc_times));

  float gpu_ms;
  gettimeofday(&tval_after, NULL);
  hipEventElapsedTime(&gpu_ms, start, stop);
  printf("%f msec\n", gpu_ms);
}

static void
calculate(int _number_of_curve, int _N, bool _write)
{
  Config config;

  config.R = 0.020;
  config.N = _N;
  config.TC = _N;
  config.initial_temperature = 850.0;
  config.final_temperature = 20.0;
  config.cooling_curve_N = 600;
  config.htc_N = 200;

  hipHostMalloc( (void**)&config.htc_times, config.htc_N * sizeof(PREC) );
  hipHostMalloc( (void**)&config.c1pc1p2i, config.N * sizeof(PREC) );
  hipHostMalloc( (void**)&config.c1mc1p2i, config.N * sizeof(PREC) );
  hipHostMalloc( (void**)&config.htc_values, _number_of_curve * config.htc_N * sizeof(PREC) );
  hipHostMalloc( (void**)&config.cooling_curve_times, config.cooling_curve_N * sizeof(PREC) );
  hipHostMalloc( (void**)&config.cooling_curve_values, _number_of_curve * config.cooling_curve_N * sizeof(PREC) );

  read_htc(&config,"./test_data/htc_time.dat","./test_data/htc_value.dat");
  reproduce_cooling_curve(&config);
  reproduce_htc_values(&config, _number_of_curve);

  CalculateWorker( _number_of_curve, &config);

  if (_write) write_result(&config, _number_of_curve);

  hipHostFree(config.cooling_curve_values);
  hipHostFree(config.htc_values);
  hipHostFree(config.htc_times);
  hipHostFree(config.cooling_curve_times);
  hipHostFree(config.c1pc1p2i);
  hipHostFree(config.c1mc1p2i);
}

int
main(int argc, char **argv)
{
  calculate(1, 32, false);
  calculate(32, 96, false);
  calculate(1000, 96, true);
  calculate(10000, 96, true);
}

// nvcc -arch=sm_75 -O3 ./gpu.cu -o gpu
// nvcc -arch=sm_75 -O3 -v ./gpu.cu -o gpu
// nvcc -arch=sm_75 -O3 --resource-usage ./gpu.cu -o gpu
// nvcc -arch=sm_75 -O3 --maxrregcount=32 --resource-usage ./gpu.cu -o gpu
// nvcc -O0 -lineinfo ./gpu.cu --maxrregcount=64  -o gpu
